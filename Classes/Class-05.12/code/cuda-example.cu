
#include <hip/hip_runtime.h>
#include <iostream>

//checking for errors with memory: cuda-memcheck ./lab1
//profiling of tasks and their time: nvprof ./lab1 profiling 

//global kernel
//this function - kernel, will run the same way on all the threads
__global__ void add(float *d_vec1, float *d_vec2, int N)
{
    //all threads in the block have indexes - here we are taking these indices
    //all the threads will index from (n*i), where i is the number of the block, n is the length of all blocks
    
    //block id for accessing the right memory part
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < N)
    {
        //h is used for device variables (d_name)
        d_vec1[tid] = d_vec1[tid] + d_vec2[tid];
    }
}

int main(int argc , char **argsv)
{
    //a big table for that GPU would be useful
    //two tables of 1048576 (2 ^20) elements
    const int N = 1 << 20;
    float *h_vec1 = new float[N], *h_vec2 = new float[N];

    //d is used for device variables (d_name)
    float *d_vec1, *d_vec2;

    //we're allocating memory on the card
    hipMalloc(&d_vec1, N * sizeof(float));
    hipMalloc(&d_vec2, N * sizeof(float));

    //we're filling our two tables
    for (int i=0; i < N; i++)
    {
        h_vec1[i] = 1.0f;
        h_vec2[i] = 2.0f;
    }

    
    hipMemcpy(d_vec1, h_vec1, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vec2, h_vec2, N * sizeof(float), hipMemcpyHostToDevice);

    //the same but on CPU - iterativly
    for(int i = 0; i< N; i++)
    {
        //h_vec1[i] += h_vec2[i];
    }

    dim3 num_threads(1024);
    dim3 num_blocks(N / 1024);

    //adding on GPU by the kernel
    //max to 2048 or 1024 threads in one block
    //we copy the data to the card as the card has no access to the RAM
    add<<<num_blocks, num_threads>>>(d_vec1, d_vec2, N);

    //we're synchronizing all the threads
    //to this point all the counting must have ended - we will be taking the data from the GPU
    hipDeviceSynchronize();

    //taking data from the card to our device
    //cudaMemcpy has cudaDeviceSynchronize inside it (it will wait on its own) - but I kept it for you to remember that it is not a given. Always check if your next function after what you're doing on GPU will synchronize.
    hipMemcpy(h_vec1, d_vec1, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_vec2, d_vec2, N * sizeof(float), hipMemcpyDeviceToHost);

    //checking results
    for(int i=0; i < N; i++)
    {
        if(h_vec1[i] != 3.0f)
        {
            std::cout << "Incorrect result" << std::endl;
            return 1;
        }
        else
        {
            std::cout << "Correct result" << std::endl;
            return 1; 
        }
    }

    //cleaning the memory on host
    delete[] h_vec1;
    delete[] h_vec2;

    //cleaning the memory on device
    hipFree(d_vec1);
    hipFree(d_vec2);
} 